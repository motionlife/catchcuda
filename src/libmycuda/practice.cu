#include "hip/hip_runtime.h"
#include "practice.h"
#include <hip/hip_runtime.h>
#include <hiprand.h>
#include <>

#include <chrono>
#include <iostream>
using namespace std;

__global__ void addTen(float *d, int count) {
  int tpb = blockDim.x * blockDim.y * blockDim.z;
  int tpib = threadIdx.x + blockDim.x * threadIdx.y +
             blockDim.x * blockDim.y * threadIdx.z;
  int bpg =
      blockIdx.x + gridDim.x * blockIdx.y + gridDim.x * gridDim.y * blockIdx.z;
  int tid = bpg * tpb + tpib;
  if (tid < count)
    d[tid] += 10;
}

int map_fun(int count) {
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  hiprandSetPseudoRandomGeneratorSeed(gen, time(nullptr));

  //  const int count = 123456;
  int size = count * sizeof(float);
  float *d;
  float h[count];
  hipMalloc(&d, size);
  hiprandGenerateUniform(gen, d, count);

  dim3 block(8, 8, 8);
  dim3 grid(16, 16);

  addTen<<<grid, block>>>(d, count);
  hipMemcpy(h, d, size, hipMemcpyDeviceToHost);
  hipFree(d);
  for (auto n : h) {
    cout << n << endl;
  }
  return 0;
}

int gather_fun(int count){
  hiprandGenerator_t gen;
  hiprandCreateGenerator(&gen, HIPRAND_RNG_PSEUDO_MTGP32);
  cout << count << endl;
  return 9;
}